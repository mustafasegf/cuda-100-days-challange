
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdlib>
#include <iostream>
#include <cassert>
#include <iomanip>

#define eps 1e-2f

__global__ void vector_add_kernel(float *a, float *b, float *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= n || col >= n)
    {
        return;
    }

    float sum = 0.0;

    for (size_t i = 0; i < n; i++)
    {
        sum += a[row * n + i] * b[i * n + col];
    }

    c[row * n + col] = sum;
}

void generate_vector(float *m, int n)
{
    for (size_t i = 0; i < n; i++)
    {
        for (size_t j = 0; j < n; j++)
        {
            m[i * n + j] = static_cast<float>(rand()) / RAND_MAX * 100.0f;
        }
    }
}

void print_assert_matrix(float *matrix_gpu, float *matrix_cpu, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            float gpu_val = matrix_gpu[i * n + j];
            float cpu_val = matrix_cpu[i * n + j];
            float diff = fabs(gpu_val - cpu_val);

            std::cout << std::fixed << std::setprecision(6);
            std::cout << "i=" << std::setw(2) << i
                      << " j=" << std::setw(2) << j
                      << " | GPU: " << std::setw(10) << gpu_val
                      << " | CPU: " << std::setw(10) << cpu_val
                      << " | DIFF: " << std::setw(10) << diff;

            if (diff >= eps)
            {
                std::cout << "  <-- ASSERT FAILED";
                std::cout << std::endl;
                assert(diff < eps);
            }

            std::cout << std::endl;
        }
    }
}

void vector_add_cpu(float *a, float *b, float *c, int n)
{
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
        {
            float sum = 0.0f;
            for (int k = 0; k < n; k++)
                sum += a[i * n + k] * b[k * n + j];
            c[i * n + j] = sum;
        }
}

int main()
{
    srand(0);
    int n = 16;
    size_t size = n * n * sizeof(float);

    float *a_h = (float *)malloc(size);
    float *b_h = (float *)malloc(size);
    float *c_h = (float *)malloc(size);
    float *c_ref = (float *)malloc(size);

    generate_vector(a_h, n);
    generate_vector(b_h, n);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **)&a_d, size);
    hipMalloc((void **)&b_d, size);
    hipMalloc((void **)&c_d, size);

    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

    size_t blockSize = 16;
    dim3 dimGrid((n + blockSize - 1) / blockSize, (n + blockSize - 1) / blockSize);
    dim3 dimBlock(blockSize, blockSize);

    vector_add_kernel<<<dimGrid, dimBlock>>>(a_d, b_d, c_d, n);
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    vector_add_cpu(a_h, b_h, c_ref, n);

    print_assert_matrix(c_h, c_ref, n);
}